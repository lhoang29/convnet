#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>

#include <layer_kernels.cuh>

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
                            const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp);
        
        /*
         * Compute the probability of guessing the correct case if you take the most-probable label.
         * 
         * This is done like this:
         * 
         * - If the most probable label is not equal to the true label, then the probability is zero.
         * - Otherwise, the probability is 1 / (number of labels whose probability is equal to the maximum).
         * 
         * This is certainly overkill -- in practice, it's just about impossible for two labels to get assigned
         * maximum probability. But it's a safety measure to prevent over-estimating your accuracy.
         * Though it could never happen in reality. Well it could. But it wouldn't. Cool?
         */
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dy_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregCostGrad(float* y_l, float* labels, float* dE_dy_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * (label == ty);
        v = __fdividef(v, y_l[tidx]);
        if (add) {
            dE_dy_l[tidx] += v;
        } else {
            dE_dy_l[tidx] = v;
        }
    }
}

/*
 * dE_dy_l: (numOut, numCases)
 * y_l:     (numOut, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kSoftmaxGrad(float* dE_dy_l, float* y_l, float* dE_dx_l, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        float v = 0;
        for (int j = 0; j < numOut; j++) {
            v += dE_dy_l[j * numCases + tx] * ((j == ty) - y_l[j * numCases + tx]);
        }
        v *= y_l[tidx];
        
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

/*
 * E = -log(y_t)
 * y_l:     (numOut, numCases)
 * labels:  (1, numCases)
 * 
 * dE_dx_l: (numOut, numCases)
 */
template <bool add>
__global__ void kLogregSoftmaxGrad(float* y_l, float* labels, float* dE_dx_l, const int numCases,
                                 const int numOut, const float gradCoeff) {
    const int tx = blockIdx.x * LOGREG_GRAD_THREADS_X + threadIdx.x;
    const int ty = blockIdx.y * LOGREG_GRAD_THREADS_Y + threadIdx.y;
    const int tidx = ty * numCases + tx;
    
    if (ty < numOut && tx < numCases) {
        const int label = int(labels[tx]);
        float v = gradCoeff * ((label == ty) - y_l[tidx]);
        if (add) {
            dE_dx_l[tidx] += v;
        } else {
            dE_dx_l[tidx] = v;
        }
    }
}

template <int B_X, bool add>
__global__ void kEltwiseMaxGrad(float* actGrad, float* input, float* output, float* target,
                                const int numElements) {
    for (int i = B_X * blockIdx.x + threadIdx.x; i < numElements; i += B_X * gridDim.x) {
        if (add) {
            target[i] += actGrad[i] * (output[i] == input[i]);
        } else {
            target[i] = actGrad[i] * (output[i] == input[i]);
        }
    }
}

void computeEltwiseMaxGrad(NVMatrix& actGrad, NVMatrix& input, NVMatrix& output, NVMatrix& target, bool add) {
    assert(actGrad.isContiguous());
    assert(output.isContiguous());
    assert(input.isContiguous());
    assert(actGrad.isSameDims(input));
    assert(actGrad.isSameDims(output));
    
    dim3 blocks(DIVUP(actGrad.getNumElements(), 128));
    dim3 threads(128);
    if (add) {
        assert(actGrad.isSameDims(target));
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), true>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, true><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    } else {
        target.resize(actGrad);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kEltwiseMaxGrad<128), false>, hipFuncCachePreferL1);
        kEltwiseMaxGrad<128, false><<<blocks, threads>>>(actGrad.getDevData(), input.getDevData(), output.getDevData(), target.getDevData(), actGrad.getNumElements());
    }
    
    getLastCudaError("computeEltwiseMaxGrad: Kernel execution failed");
}

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases)
 */
void computeLogregCost(NVMatrix& labels, NVMatrix& probs, NVMatrix& labelLogProbs_out, NVMatrix& correctProbs_out) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    
    NVMatrix& maxProbs = probs.max(0);
    
    labelLogProbs_out.resize(1, numCases);
    correctProbs_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLogregCost), hipFuncCachePreferL1);
    kLogregCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     labelLogProbs_out.getDevData(), correctProbs_out.getDevData(),
                                     numCases, numOut);
    getLastCudaError("computeLogregCost: Kernel execution failed");
//    hipDeviceSynchronize();
    delete &maxProbs;
}

void computeLogregGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregCostGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregCostGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    getLastCudaError("computeLogregGrad: Kernel execution failed");
}

void computeSoftmaxGrad(NVMatrix& acts, NVMatrix& actsGrad, NVMatrix& target, bool add) {
    int numCases = acts.getLeadingDim();
    int numOut = acts.getFollowingDim();

    assert(acts.isSameDims(actsGrad));
    assert(acts.isContiguous());
    assert(actsGrad.isContiguous());
    assert(target.isContiguous());
    assert(acts.isTrans());
    assert(actsGrad.isTrans());

    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(acts);
        kSoftmaxGrad<false><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    } else {
        kSoftmaxGrad<true><<<blocks, threads>>>(actsGrad.getDevData(), acts.getDevData(), target.getDevData(), numCases, numOut);
    }
    getLastCudaError("computeSoftmaxGrad: Kernel execution failed");
}

void computeLogregSoftmaxGrad(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getLeadingDim(); 
    int numOut = probs.getFollowingDim(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(probs.isTrans());
    
    dim3 threads(LOGREG_GRAD_THREADS_X, LOGREG_GRAD_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRAD_THREADS_X), DIVUP(numOut, LOGREG_GRAD_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregSoftmaxGrad<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregSoftmaxGrad<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    }

    getLastCudaError("computeLogregSoftmaxGrad: Kernel execution failed");
}



///////// Modif Q : aggregation
//

__global__ void kAggForward(float* inputs, float* outputs, int* aggArray, const int arrlength, const int numCases,
                                 const int numClasses, int* chosenParent) {
	const int tidx = threadIdx.x;
	
	if (tidx<numCases) {
		inputs 		+= tidx*numClasses;
		outputs 	+= tidx*numClasses;
		chosenParent 	+= tidx*numClasses;

		int iter=0;
		int k, cls, numparents, maxparent;
		float outvalue, parentval;
		int parents[3];
		float parentvalues[3];

		for (cls=0;cls<numClasses;cls++)
		{
			numparents=aggArray[iter];
			iter++;
			outvalue=inputs[cls];
			maxparent=-1;
			if (numparents>0) {
				// get score values from parents + indices
				for (k=0;k<numparents;k++)
				{
					parents[k]=aggArray[iter+k];
					parentvalues[k]=inputs[parents[k]];
				}

				// find the max one
				maxparent=parents[0];
				parentval=parentvalues[0];
				for (k=1;k<numparents;k++)
				{
					if (parentvalues[k] > parentval)
					{
						parentval=parentvalues[k];
						maxparent=parents[k];
					}
				}
				outvalue+=parentval;
			}
			chosenParent[cls]=maxparent;
			outputs[cls]=outvalue;
			iter=iter+numparents;
		
		}
	}
}



__global__ void kAggBackward(float* gradOutputs, float* gradInputs, const int numCases,
                                 const int numClasses, int* chosenParent) {
	const int tidx = threadIdx.x;
	if (tidx<numCases) {

		gradOutputs 	+= tidx*numClasses;
		gradInputs 	+= tidx*numClasses;
		chosenParent 	+= tidx*numClasses;
	
		int k, parent;

		for (k=numClasses-1; k>=0; k--) {
			parent=chosenParent[k];
			if (parent>-1) {
				gradInputs[parent] += gradOutputs[k];
			}
		}

	}
}


void computeAggForward(NVMatrix& inputs, NVMatrix& outputs, int* aggArray, const int _length, int* chosenParent)
{
	const int numClasses=inputs.getNumCols();
	const int numCases=inputs.getNumRows();
	outputs.resize(inputs.getNumRows(),inputs.getNumCols());
	dim3 threads(128,1);
	dim3 blocks (1,1);
	kAggForward<<<blocks,threads>>>(inputs.getDevData(), outputs.getDevData(), aggArray, _length, numCases, numClasses, chosenParent);
        getLastCudaError("computeAggForward: Kernel execution failed");
}

void computeAggBackward(NVMatrix& gradOut, NVMatrix& gradIn, int* chosenParent)
{
	gradOut.copy(gradIn);
	const int numClasses=gradOut.getNumCols();
	const int numCases=gradOut.getNumRows();
	dim3 threads(128,1);
	dim3 blocks (1,1);
	kAggBackward<<<blocks,threads>>>(gradOut.getDevData(), gradIn.getDevData(), numCases, numClasses, chosenParent);
        getLastCudaError("computeAggBackward: Kernel execution failed");
}
